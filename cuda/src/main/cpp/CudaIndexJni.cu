

  #include <stdio.h>
  #include <math.h>
  #include "CudaIndexJni.h"
  
  #include <thrust/transform.h>
  #include <thrust/functional.h>
  #include <thrust/host_vector.h>
  #include <thrust/device_vector.h>
  #include <vector>
  #include <sys/time.h>
  
  using namespace std;
  using namespace thrust;

  long ms () {
      struct timeval tp;
      gettimeofday(&tp, NULL);
      return tp.tv_sec * 1000 + tp.tv_usec / 1000; //get current timestamp in milliseconds
  }
  
  long T;
  long P;

  host_vector<int> startPositionsCpu;
  device_vector<int> docIdsGpu;
  device_vector<float> partialScoresGpu;

  JNIEXPORT jint JNICALL Java_CudaIndexJni_initIndex
  (JNIEnv *env, jobject jobj, jintArray docIds, jfloatArray partialScores, jintArray startPositions) {
      jsize len = env->GetArrayLength(startPositions);
      jsize numPostings = env->GetArrayLength(docIds);
      vector<int> docs (numPostings);
      env->GetIntArrayRegion( docIds, 0, numPostings, &docs[0] );
      vector<float> scores (numPostings);
      env->GetFloatArrayRegion( partialScores, 0, numPostings, &scores[0] );
      int *starts = env->GetIntArrayElements(startPositions, NULL);
  
      T = len;
      P = numPostings;

      for (int i=0; i<T; i++) {
        startPositionsCpu.push_back(starts[i]);
      }
      docIdsGpu = docs;
      partialScoresGpu = scores;
      // Copy the vectors to the device
      /*host_vector<int> cpu_docs(N);
      for (int i=0; i<N; i++) cpu_docs[i] = docs[i];
      gpu_docs = cpu_docs;
  
      host_vector<float> cpu_lats(N);
      for (int i=0; i<N; i++) cpu_lats[i] = lats[i];
      gpu_lats = cpu_lats;
  
      host_vector<float> cpu_lngs(N);
      for (int i=0; i<N; i++) cpu_lngs[i] = lngs[i];
      gpu_lngs = cpu_lngs;*/

      /*startPositionsGpu = starts;
      docIdsGpu = docs;
      partialsScoresGpu = scores;
  
      hipDeviceSynchronize();
      */return T;
  }

JNIEXPORT jobject JNICALL Java_CudaIndexJni_getScores
  (JNIEnv *env, jobject jobj, jintArray terms)
  {
    jsize Q = env->GetArrayLength(terms);
    vector<int> queryTerms (Q);
    env->GetIntArrayRegion( terms, 0, Q, &queryTerms[0] );

    /*for (int i=0; i<T; i++) {
        cout<<startPositionsCpu[i]<<", ";
    } cout<<endl;
  
    for (int i=0; i<P; i++) {
        cout<<docIdsGpu[i]<<"="<<partialScoresGpu[i]<<", ";
    }
    cout<<endl;*/
    cout<<"Initialized CUDA with terms "<<T<<" and query terms "<<Q<<endl;
    cout<<"Postings: "<<P<<endl;
      /*long timer = ms();
  
      // Actual scoring/sorting on device
      device_vector<float> gpu_distances(N);
      thrust::transform(gpu_lats.begin(), gpu_lats.end(), gpu_lngs.begin(), gpu_distances.begin(), geodist(lat, lng)  );
      hipDeviceSynchronize();
      device_vector<int> docIds = gpu_docs;
      cout<<"Transformation applied: "<<endl;
      thrust::sort_by_key(gpu_distances.begin(), gpu_distances.end(), docIds.begin());
      hipDeviceSynchronize();
      cout<<"Sorting done: "<<endl;
  
      int   *docs      = (int*)malloc(N*4*2);
      float *distances = &((float*)docs)[N];
  
      thrust::copy(docIds.begin(), docIds.end(), docs);
      thrust::copy(gpu_distances.begin(), gpu_distances.end(), distances);
  
      jobject directBuffer = env->NewDirectByteBuffer((void*)docs, N*2*4);
      cout<<"Cuda After array copy total time: "<<ms()-timer<<endl;
  
      return directBuffer;*/

      int mergedSize = 0;
      for (int q=0; q<queryTerms.size(); q++) {
          mergedSize += startPositionsCpu[queryTerms[q]+1]-startPositionsCpu[queryTerms[q]];
      }
      device_vector<int> mergedDocIds(mergedSize);
      device_vector<float> mergedPartialScores(mergedSize);
      device_vector<float> reducedValues(mergedSize);
      device_vector<int>   reducedKeys(mergedSize);
  
      int pos = 0;
      for (int q=0; q<queryTerms.size(); q++) {
          int n = startPositionsCpu[queryTerms[q]+1]-startPositionsCpu[queryTerms[q]];
          copy_n(device, docIdsGpu.begin() + (startPositionsCpu[queryTerms[q]]), n, mergedDocIds.begin() + pos);
          copy_n(device, partialScoresGpu.begin() + (startPositionsCpu[queryTerms[q]]), n, mergedPartialScores.begin() + pos);
  
          pos += n;
      }
  
      thrust::sort_by_key(mergedDocIds.begin(), mergedDocIds.end(), mergedPartialScores.begin());
      thrust::pair<device_vector<int>::iterator,device_vector<float>::iterator > p = reduce_by_key(device,
                    mergedDocIds.begin(), mergedDocIds.end(),
                    mergedPartialScores.begin(),
                    reducedKeys.begin(),
                    reducedValues.begin());
      thrust::sort_by_key(reducedValues.begin(), p.second, reducedKeys.begin(), thrust::greater<float>());
      //t.stop(); cout<<"Time: "<<t.elapsed()/1000000.0<<endl;
  
      cout<<"Size of merged docid: "<<mergedDocIds.size()<<endl;

      cout<<"(CUDA) Doc "<<reducedKeys[0]<<": "<<"score="<<reducedValues[0]<<endl;
      cout<<"(CUDA) Doc "<<reducedKeys[1]<<": "<<"score="<<reducedValues[1]<<endl;
      cout<<"(CUDA) Doc "<<reducedKeys[2]<<": "<<"score="<<reducedValues[2]<<endl;
  
      return NULL;
  }
  
  
  
  
  