#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "GpuGeoDist.h"

#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector>
#include <sys/time.h>

using namespace std;
using namespace thrust;

struct geodist {
	const float refX;
	const float refY;

	geodist(float _x, float _y): refX(_x), refY(_y) {}

	__host__ __device__ float operator()(float &x, float &y) const {
        return sqrt((x - refX) * (x-refX) + (y-refY)*(y-refY));
    }
};

long ms () {
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return tp.tv_sec * 1000 + tp.tv_usec / 1000; //get current timestamp in milliseconds
}

int *docs;
float *lats;
float *lngs;
long N;

device_vector<int> gpu_docs;
device_vector<float> gpu_lats;
device_vector<float> gpu_lngs;

JNIEXPORT jint JNICALL Java_GpuGeoDist_initIndex(JNIEnv *env, jobject obj, jintArray docIdsArray, jfloatArray latsArray, jfloatArray lngsArray) {
    jsize len = env->GetArrayLength(docIdsArray);
    docs = env->GetIntArrayElements(docIdsArray, NULL);
    lats = env->GetFloatArrayElements(latsArray, NULL);
    lngs = env->GetFloatArrayElements(lngsArray, NULL);

    N = len;

    // Copy the vectors to the device
    host_vector<int> cpu_docs(N);
    for (int i=0; i<N; i++) cpu_docs[i] = docs[i];
    gpu_docs = cpu_docs;

    host_vector<float> cpu_lats(N);
    for (int i=0; i<N; i++) cpu_lats[i] = lats[i];
    gpu_lats = cpu_lats;

    host_vector<float> cpu_lngs(N);
    for (int i=0; i<N; i++) cpu_lngs[i] = lngs[i];
    gpu_lngs = cpu_lngs;

    hipDeviceSynchronize();
    return N;
}


JNIEXPORT jobject JNICALL Java_GpuGeoDist_findNearest(JNIEnv *env, jobject obj, jfloat lat, jfloat lng)
{

    long timer = ms();

	// Actual scoring/sorting on device
    device_vector<float> gpu_distances(N);
    thrust::transform(gpu_lats.begin(), gpu_lats.end(), gpu_lngs.begin(), gpu_distances.begin(), geodist(lat, lng)  );
    hipDeviceSynchronize();
    device_vector<int> docIds = gpu_docs;
    cout<<"Transformation applied: "<<endl;
    thrust::sort_by_key(gpu_distances.begin(), gpu_distances.end(), docIds.begin());
    hipDeviceSynchronize();
    cout<<"Sorting done: "<<endl;

    int   *docs      = (int*)malloc(N*4*2);
    float *distances = &((float*)docs)[N];

    thrust::copy(docIds.begin(), docIds.end(), docs);
    thrust::copy(gpu_distances.begin(), gpu_distances.end(), distances);

    jobject directBuffer = env->NewDirectByteBuffer((void*)docs, N*2*4);
    cout<<"Cuda After array copy total time: "<<ms()-timer<<endl;

    return directBuffer;
}




