

  #include <stdio.h>
  #include <math.h>
  #include "CudaIndexJni.h"
  
/*  #include <thrust/transform.h>
  #include <thrust/functional.h>
  #include <thrust/host_vector.h>
  #include <thrust/device_vector.h>*/
  #include <vector>
  #include <sys/time.h>
  
  using namespace std;
//  using namespace thrust;

  long ms () {
      struct timeval tp;
      gettimeofday(&tp, NULL);
      return tp.tv_sec * 1000 + tp.tv_usec / 1000; //get current timestamp in milliseconds
  }
  
  long N;
  
  /*host_vector<int> startPositionsGpu
  device_vector<int> docIdsGpu;
  device_vector<float> partialScoresGpu;*/

  JNIEXPORT jint JNICALL Java_CudaIndexJni_initIndex
  (JNIEnv *env, jobject jobj, jintArray docIds, jfloatArray partialScores, jintArray startPositions) {
      jsize len = env->GetArrayLength(startPositions);
      int *docs = env->GetIntArrayElements(docIds, NULL);
      float *scores = env->GetFloatArrayElements(partialScores, NULL);
      int *starts = env->GetIntArrayElements(startPositions, NULL);
  
      N = len;
  
      // Copy the vectors to the device
      /*host_vector<int> cpu_docs(N);
      for (int i=0; i<N; i++) cpu_docs[i] = docs[i];
      gpu_docs = cpu_docs;
  
      host_vector<float> cpu_lats(N);
      for (int i=0; i<N; i++) cpu_lats[i] = lats[i];
      gpu_lats = cpu_lats;
  
      host_vector<float> cpu_lngs(N);
      for (int i=0; i<N; i++) cpu_lngs[i] = lngs[i];
      gpu_lngs = cpu_lngs;*/

      /*startPositionsGpu = starts;
      docIdsGpu = docs;
      partialsScoresGpu = scores;
  
      hipDeviceSynchronize();
      */return N;
  }

JNIEXPORT jobject JNICALL Java_CudaIndexJni_getScores
  (JNIEnv *env, jobject jobj, jintArray terms)
  {
  
      /*long timer = ms();
  
      // Actual scoring/sorting on device
      device_vector<float> gpu_distances(N);
      thrust::transform(gpu_lats.begin(), gpu_lats.end(), gpu_lngs.begin(), gpu_distances.begin(), geodist(lat, lng)  );
      hipDeviceSynchronize();
      device_vector<int> docIds = gpu_docs;
      cout<<"Transformation applied: "<<endl;
      thrust::sort_by_key(gpu_distances.begin(), gpu_distances.end(), docIds.begin());
      hipDeviceSynchronize();
      cout<<"Sorting done: "<<endl;
  
      int   *docs      = (int*)malloc(N*4*2);
      float *distances = &((float*)docs)[N];
  
      thrust::copy(docIds.begin(), docIds.end(), docs);
      thrust::copy(gpu_distances.begin(), gpu_distances.end(), distances);
  
      jobject directBuffer = env->NewDirectByteBuffer((void*)docs, N*2*4);
      cout<<"Cuda After array copy total time: "<<ms()-timer<<endl;
  
      return directBuffer;*/
      return NULL;
  }
  
  
  
  
  